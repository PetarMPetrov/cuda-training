
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <cstdlib>
#include <chrono>

int A[3][4] = {1,1,1,1, 1,1,1,1, 2,2,2,2};

int B[4][2] = {5, 10, 5, 10, 5, 10, 5, 10};

int C[3][2]; // = { 0, 0, 0 ,0, 0, 0};

template<std::size_t NROWS, std::size_t NCOLS, std::size_t VLEN>
__global__ void matMulGPU(int (&mA)[NROWS][VLEN], int (&mB)[VLEN][NCOLS], int (&mC)[NROWS][NCOLS])
{
    int op_id = blockIdx.x * blockDim.x + threadIdx.x;
    int row = op_id / NCOLS;
    int col = op_id % NCOLS;
    if (op_id < NCOLS * NROWS)
    {
        for(int k = 0; k < VLEN; k++)
        {
            mC[row][col] += mA[row][k] * mB[k][col];
        }
    }
}

int main()
{
    const unsigned int nrows = 3;
    const unsigned int ncols = 2;
    const unsigned int vlen  = 4;

    int _A[nrows][vlen];
    int _B[vlen][ncols];
    int _C[nrows][ncols];

    // Initialize _A
    for (int i = 0; i < nrows; i++)
    {
        for(int j = 0; j < vlen; j++)
        {
            _A[i][j] = std::rand()%10;
        }
    }
    // Initialize _B
    for (int i = 0; i < vlen; i++)
    {
        for(int j = 0; j < ncols; j++)
        {
            _B[i][j] = std::rand()%10;
        }
    }
    // Initialize _C
    for (int i = 0; i < nrows; i++)
    {
        for(int j = 0; j < ncols; j++)
        {
            _C[i][j] = 0;
        }
    }

    // Matrix Multiplication
    auto start = std::chrono::high_resolution_clock::now();
    // Make sure nrows * ncols is exactly divisible by nBlocks
    const std::size_t nBlocks = 1;
    int mA[nrows][vlen], mB[vlen][ncols], mC[nrows][ncols];
    std::size_t sizeA = sizeof(int) * nrows * vlen;
    std::size_t sizeB = sizeof(int) * ncols * vlen;
    std::size_t sizeC = sizeof(int) * nrows * ncols;
    hipMalloc((void**)&mA, sizeA);
    hipMalloc((void**)&mB, sizeB);
    hipMalloc((void**)&mC, sizeC);
    hipMemcpy(mA, A, sizeA, hipMemcpyHostToDevice); 
    hipMemcpy(mB, B, sizeB, hipMemcpyHostToDevice); 
    matMulGPU<nrows,ncols,vlen><<<nBlocks, nrows * ncols / nBlocks>>>(mA, mB, mC);
    hipMemcpy(mC, C, sizeC, hipMemcpyDeviceToHost); 
    hipFree(mA);
    hipFree(mB);
    hipFree(mC);
    
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "MatMul finished in " << duration.count() << " microsec" << std::endl;
    //matMulCPU(_A, _B, _C, nrows, ncols, vlen);
/*    for(int i = 0; i < nrows; i++)
    {
        for(int j = 0; j < ncols; j++)
        {
            for(int k = 0; k < vlen; k++)
            {
                _C[i][j] += _A[i][k] * _B[k][j];
            }
        }
    }
*/
    for(int i = 0; i < 6; i++){
        const int row = i/2;
        const int col = i%2;
        std::cout << "C[" << row << "][" << col << "] = " << C[row][col] <<std::endl;
    }

    // Print small Matrices
    if(nrows * ncols < 20 && false)
    {
        for(int i = 0; i < nrows * vlen; i++){
            const int row = i/vlen;
            const int col = i%vlen;
            std::cout << "A[" << row << "][" << col << "] = " << _A[row][col] <<std::endl;
        }
        for(int i = 0; i < vlen * ncols; i++){
            const int row = i/ncols;
            const int col = i%ncols;
            std::cout << "B[" << row << "][" << col << "] = " << _B[row][col] <<std::endl;
        }
        for(int i = 0; i < nrows * ncols; i++){
            const int row = i/ncols;
            const int col = i%ncols;
            std::cout << "C[" << row << "][" << col << "] = " << _C[row][col] <<std::endl;
        }
    }
    return 0;
}
